#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s

// expected-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

__device__ void foo() {
  extern __shared__ int x; // exxpected-error {{__shared__ variable 'x' cannot be 'extern'}}
  extern __shared__ int arr[];  // ok
  extern __shared__ int arr0[0]; // exxpected-error {{__shared__ variable 'arr0' cannot be 'extern'}}
  extern __shared__ int arr1[1]; // exxpected-error {{__shared__ variable 'arr1' cannot be 'extern'}}
  extern __shared__ int* ptr ; // exxpected-error {{__shared__ variable 'ptr' cannot be 'extern'}}
}

__host__ __device__ void bar() {
  extern __shared__ int arr[];  // ok
  extern __shared__ int arr0[0]; // exxpected-error {{__shared__ variable 'arr0' cannot be 'extern'}}
  extern __shared__ int arr1[1]; // exxpected-error {{__shared__ variable 'arr1' cannot be 'extern'}}
  extern __shared__ int* ptr ; // exxpected-error {{__shared__ variable 'ptr' cannot be 'extern'}}
}

extern __shared__ int global; // exxpected-error {{__shared__ variable 'global' cannot be 'extern'}}
extern __shared__ int global_arr[]; // ok
extern __shared__ int global_arr1[1]; // exxpected-error {{__shared__ variable 'global_arr1' cannot be 'extern'}}
